#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 4000  // Tamanho da grade
#define T 500   // Número de iterações no tempo
#define D 0.1   // Coeficiente de difusão
#define DELTA_T 0.01
#define DELTA_X 1.0

// Kernel CUDA para calcular a nova matriz de concentração
__global__ void diff_eq_kernel(double *C, double *C_new, double *difmedio, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        C_new[i * n + j] = C[i * n + j] + D * DELTA_T * (
            (C[(i + 1) * n + j] + C[(i - 1) * n + j] +
             C[i * n + (j + 1)] + C[i * n + (j - 1)] -
             4.0 * C[i * n + j]) / (DELTA_X * DELTA_X)
        );

        atomicAdd(difmedio, fabs(C_new[i * n + j] - C[i * n + j]));
    }
}

// Função principal
int main() {
    size_t size = N * N * sizeof(double);

    // Alocar memória no host (CPU)
    double *C = (double *)malloc(size);
    double *C_new = (double *)malloc(size);

    if (C == NULL || C_new == NULL) {
        fprintf(stderr, "Falha na alocação de memória no host\n");
        return 1;
    }

    // Inicializar as matrizes no host
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C[i * N + j] = 0.0;
            C_new[i * N + j] = 0.0;
        }
    }
    C[N / 2 * N + N / 2] = 1.0; // Concentração inicial no centro

    // Alocar memória no dispositivo (GPU)
    double *d_C, *d_C_new, *d_difmedio;
    hipMalloc((void **)&d_C, size);
    hipMalloc((void **)&d_C_new, size);
    hipMalloc((void **)&d_difmedio, sizeof(double));

    // Copiar dados do host para o dispositivo
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C_new, C_new, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Loop de iterações no tempo
    double start_time = clock();
    for (int t = 0; t < T; t++) {
        double difmedio = 0.0;
        hipMemcpy(d_difmedio, &difmedio, sizeof(double), hipMemcpyHostToDevice);

        // Executar o kernel
        diff_eq_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_C_new, d_difmedio, N);
        hipDeviceSynchronize();

        // Trocar os ponteiros
        double *temp = d_C;
        d_C = d_C_new;
        d_C_new = temp;

        // Obter difmedio do dispositivo
        hipMemcpy(&difmedio, d_difmedio, sizeof(double), hipMemcpyDeviceToHost);

        if ((t % 100) == 0) {
            printf("Interação %d - diferença = %g\n", t, difmedio / ((N - 2) * (N - 2)));
        }
    }
    double end_time = clock();

    // Copiar os resultados finais para o host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Concentração final no centro: %f\n", C[N / 2 * N + N / 2]);
    printf("Tempo de execução: %f segundos\n", (end_time - start_time) / CLOCKS_PER_SEC);

    // Liberar memória no dispositivo e no host
    hipFree(d_C);
    hipFree(d_C_new);
    hipFree(d_difmedio);
    free(C);
    free(C_new);

    return 0;
}
